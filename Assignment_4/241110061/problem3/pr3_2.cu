#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <hip/hip_runtime.h>
#include <ctime>
#include <vector>
#include <iostream>

int threadsPerBlock = 512;  // Threads per block (modifiable)
int blocksPerGrid = 1024;   // Blocks per grid (modifiable)

void runGridLoop(double paramB[30], double paramA[120], double thresholdK);

double paramA[120];
double paramB[30];

#define cudaCheckError(ans)                                                    \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file,
            line);
    if (abort)
      exit(code);
  }
}


__global__ void computeKernel(double* output, double* paramB, double* paramC, double* paramE, int* gridSteps, long offset) {
  extern __shared__ double sharedMemory[];  
  double* comc = sharedMemory;             
  
  long index = blockIdx.x * blockDim.x + threadIdx.x;
  long tempIndex = index + offset;
  int gridPositions[10];
  
  
  for (int i = 9; i >= 0; --i) {
    gridPositions[i] = tempIndex % gridSteps[i];
    tempIndex /= gridSteps[i];
  }

  double computedValues[10];
  index *= 10;

 
  for (int i = 0; i < 10; ++i) {
    computedValues[i] = paramB[3 * i] + gridPositions[i] * paramB[3 * i + 2];
  }

  
  if (threadIdx.x < 120) {
    comc[threadIdx.x] = paramC[threadIdx.x];
  }
  __syncthreads();

  
  double comparison[10];
  for (int i = 0; i < 10; ++i) {
    comparison[i] = -comc[10 * 10 + i];
    for (int j = 0; j < 10; ++j) {
      comparison[i] += comc[j * 10 + i] * computedValues[j];
    }
    if (fabs(comparison[i]) > paramE[i]) {
      return;  
    }
  }

 
  for (int i = 0; i < 10; ++i)
    output[index + i] = computedValues[i];
}


int main() {
  int indexA = 0, indexB = 0;

  // Load data from files
  FILE* fileA = fopen("./disp.txt", "r");
  if (fileA == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }
  while (fscanf(fileA, "%lf", &paramA[indexA]) == 1) {
    indexA++;
  }
  fclose(fileA);

  FILE* fileB = fopen("./grid.txt", "r");
  if (fileB == NULL) {
    printf("Error: could not open file\n");
    return 1;
  }
  while (fscanf(fileB, "%lf", &paramB[indexB]) == 1) {
    indexB++;
  }
  fclose(fileB);

  double thresholdK = 0.3;

  // Using CUDA events for timing
  hipEvent_t start, stop;
  float elapsedTime;
  cudaCheckError(hipEventCreate(&start));
  cudaCheckError(hipEventCreate(&stop));

  cudaCheckError(hipEventRecord(start, 0));
  printf("Running program, please wait...\n");
  runGridLoop(paramB, paramA, thresholdK);
  cudaCheckError(hipEventRecord(stop, 0));
  cudaCheckError(hipEventSynchronize(stop));
  cudaCheckError(hipEventElapsedTime(&elapsedTime, start, stop));

  printf("Total time = %f seconds\n", elapsedTime / 1000.0f);

  cudaCheckError(hipEventDestroy(start));
  cudaCheckError(hipEventDestroy(stop));

  return EXIT_SUCCESS;
}
__host__ inline void runGridLoop(double paramB[30], double paramA[120], double thresholdK) {
  long pointCount = 0;
  long linesSaved = 0;

  // Allocate memory for device variables
  double *d_gridValues, *d_thresholdArray, *d_matrix;
  int *d_gridSteps;

  cudaCheckError(hipMalloc((void**)&d_gridValues, 30 * sizeof(double)));
  cudaCheckError(hipMemcpy(d_gridValues, paramB, 30 * sizeof(double), hipMemcpyHostToDevice));

  cudaCheckError(hipMalloc((void**)&d_thresholdArray, 10 * sizeof(double)));
  cudaCheckError(hipMalloc((void**)&d_matrix, 120 * sizeof(double)));
  cudaCheckError(hipMalloc((void**)&d_gridSteps, 10 * sizeof(int)));

  // Set up threshold and grid steps arrays
  double thresholdArray[10];
  int gridSteps[10];
  double matrix[12][10];
  for (int i = 0; i < 120; i++) {
    matrix[i % 12][i / 12] = paramA[i];
  }

  for (int i = 0; i < 10; ++i) {
    thresholdArray[i] = thresholdK * matrix[11][i];
    gridSteps[i] = floor((paramB[3 * i + 1] - paramB[3 * i]) / paramB[3 * i + 2]);
  }

  cudaCheckError(hipMemcpy(d_thresholdArray, thresholdArray, 10 * sizeof(double), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_matrix, matrix, 120 * sizeof(double), hipMemcpyHostToDevice));
  cudaCheckError(hipMemcpy(d_gridSteps, gridSteps, 10 * sizeof(int), hipMemcpyHostToDevice));

  long totalLoops = 1;
  for (int i = 0; i < 10; ++i) totalLoops *= gridSteps[i];

  long offset = 0;

  double *h_output, *d_output;
  h_output = (double*)malloc(threadsPerBlock * blocksPerGrid * 10 * sizeof(double));
  cudaCheckError(hipHostAlloc((void**)&d_output, threadsPerBlock * blocksPerGrid * 10 * sizeof(double), hipHostMallocDefault));

  FILE* outputFile = fopen("./results-v3_2.txt", "w");
  if (outputFile == NULL) {
    printf("Error in creating file!");
    exit(1);
  }

  while (totalLoops > 0) {
    int step = threadsPerBlock * blocksPerGrid;
    if (step > totalLoops) {
      step = totalLoops;
      if (blocksPerGrid == 1) {
        threadsPerBlock >>= 1;
      } else {
        blocksPerGrid >>= 1;
      }
    }
    
    // Launch kernel with dynamic shared memory allocation
    computeKernel<<<blocksPerGrid, threadsPerBlock, 120 * sizeof(double)>>>(d_output, d_gridValues, d_matrix, d_thresholdArray, d_gridSteps, offset);
    cudaCheckError(hipMemcpy(h_output, d_output, threadsPerBlock * blocksPerGrid * 10 * sizeof(double), hipMemcpyDeviceToHost));

    for (long j = 0; j < blocksPerGrid * threadsPerBlock; ++j) {
      if (h_output[j * 10] == 0) continue;

      ++pointCount;
      ++linesSaved;

      for (int k = 0; k < 10; ++k) {
        fprintf(outputFile, "%.6f%s", h_output[j * 10 + k], (k < 9) ? "\t" : "\n");
      }

      if (linesSaved >= 11608) {
        fclose(outputFile);
        free(h_output);
        hipFree(d_gridValues);
        hipFree(d_matrix);
        hipFree(d_thresholdArray);
        hipFree(d_gridSteps);
        hipHostFree(d_output);
        printf("\nResult points: %ld\n", pointCount);  
        return;
      }
    }

    offset += step;
    totalLoops -= step;
  }

  hipDeviceSynchronize();
  fclose(outputFile);
  printf("\nResult points: %ld\n", pointCount);

  free(h_output);
  hipFree(d_gridValues);
  hipFree(d_matrix);
  hipFree(d_thresholdArray);
  hipFree(d_gridSteps);
  hipHostFree(d_output);
}




